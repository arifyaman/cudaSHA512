#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <pthread.h>
#include <locale.h>
#include "sha512.cuh"

#define INPUT_SIZE 2
#define HASH_SIZE 64

#define THREADS 1200
#define BLOCKS 256

__global__ void sha512_kernel(unsigned char *input, unsigned char *output)
{
    SHA512_CTX ctx;
    SHA512_Init(&ctx);
    SHA512_Update(&ctx, input, INPUT_SIZE);
    SHA512_Final(output, &ctx);
}

int main()
{
    // Initialize input data
    char input[] = {"ay"};

    // Allocate memory on device for input and output
    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, INPUT_SIZE);
    hipMalloc((void **)&d_output, HASH_SIZE);

    // Copy input data from host to device
    hipMemcpy(d_input, input, sizeof(char) * INPUT_SIZE, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(dev_K512), K512, sizeof(K512), 0, hipMemcpyHostToDevice);
    // Run SHA512 kernel
    sha512_kernel<<<THREADS, BLOCKS>>>(d_input, d_output);

    // Copy output data from device to host
    unsigned char output[HASH_SIZE];
    hipMemcpy(output, d_output, HASH_SIZE, hipMemcpyDeviceToHost);

    // Print output hash
    printf("Output hash: ");
    for (int i = 0; i < HASH_SIZE; i++)
    {
        printf("%02x", output[i]);
    }
    printf("\n");

    // Free memory on device
    hipFree(dev_K512);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
