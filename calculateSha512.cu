#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <pthread.h>
#include <locale.h>
#include "sha512.cuh"

#define CHECK_TOP_N2 539858690
#define CHECK_LOW_N2 41848688

#define CHECK_TOP 25000000
#define CHECK_LOW 22222222

/* #define CHECK_TOP_N2 639858689
#define CHECK_LOW_N2 3398988

#define CHECK_TOP 50000000
#define CHECK_LOW 40000000 */


#define INPUT_SIZE 865
#define HASH_SIZE 64

#define THREADS 2000
#define BLOCKS 128

__constant__ unsigned char digits[11] = {"0123456789"};
__constant__ unsigned char input_p2[512] = {"],\"license\":{\"type\":\"gambling-virtual\",\"text\":\"Random values licensed for virtual item gambling only\",\"infoUrl\":null},\"licenseData\":null,\"userData\":\"Those numbers were generated for Crash game on INSANE.gg. The firstNumber in sequence determines the chance of x1 (instant crash). If the number is less than 40000000 (4.0% chance), then x1 will be rolled, otherwise the secondNumber in sequence is used to generate the multiplier using the formula: 1000000000 : secondNumber\",\"ticketData\":null,\"completionTime\":\""};
// 2023-04-05 18:46:58Z","serialNumber":2365441}
// 541210fe506cb91708ebd4f6c8f19525b4286ee32b338d3deb8197e71542719ce2d6a80c359abf74db192b992d4a8df10b3abc85b416a60630b5c75439ec21d4,2023-04-05 18:46:58Z,2365441
unsigned char input_p1[243] = {"{\"method\":\"generateSignedIntegers\",\"hashedApiKey\":\"DilGPW5gs5jzIsxz/8kSjH+WTLsNEQtmtrTFyq3tvgpolF41vhTwtE6iG8FT+WzdhzQZd4GBNp5q6VixloAUlA==\",\"n\":2,\"min\":0,\"max\":1000000000,\"replacement\":true,\"base\":10,\"pregeneratedRandomization\":null,\"data\":["};
unsigned char input_data[20] = {"428529917,883048505"};

__device__ void updatWithNumber(unsigned int n, unsigned char *digits, SHA512_CTX *ctx)
{

    unsigned int divisor = 1;
    while (n / divisor >= 10)
    {
        divisor *= 10;
    }
    while (divisor > 0)
    {
        SHA512_Update(ctx, &digits[(n / divisor) % 10], 1);

        divisor /= 10;
    }
}

__device__ void updateWithTimeAndSerial(unsigned char *completionTime, unsigned char *serialNumber, SHA512_CTX *ctx)
{
    SHA512_Update(ctx, completionTime, 20);
    SHA512_Update(ctx, "\",\"serialNumber\":", 17);
    SHA512_Update(ctx, serialNumber, 7);
    SHA512_Update(ctx, "}", 1);
}

__device__ void updateWithAsumedData(unsigned int seed, SHA512_CTX *ctx, unsigned int *numbers)
{
    numbers[0] = CHECK_LOW_N2 + (int)((seed + 1) / (CHECK_TOP - CHECK_LOW));
    numbers[1] = CHECK_TOP - ((seed + 1) % (CHECK_TOP - CHECK_LOW));

    updatWithNumber(numbers[0], digits, ctx);
    SHA512_Update(ctx, ",", 1);
    updatWithNumber(numbers[1], digits, ctx);
}

__global__ void sha512_kernel(SHA512_CTX *ctx, unsigned int seed, unsigned char *output, unsigned char *expectedHash, unsigned int *solution, int *blockSolution, unsigned char *completionTime, unsigned char *serialNumber)
{

    if (*blockSolution == 1)
        return;

    int i = blockIdx.x * blockDim.x * seed + threadIdx.x;

    SHA512_CTX copiedContext;
    memcpy(&copiedContext, ctx, sizeof(SHA512_CTX));

    unsigned int numbers[2];
    unsigned char foundHash[64];

    updateWithAsumedData(i, &copiedContext, numbers);

    SHA512_Update(&copiedContext, input_p2, 511);
    updateWithTimeAndSerial(completionTime, serialNumber, &copiedContext);

    SHA512_Final(foundHash, &copiedContext);

    for (int j = 0; j < 64; j++)
        if (expectedHash[j] != foundHash[j])
            return;

    *blockSolution = 1;
    memcpy(solution, numbers, sizeof(unsigned int) * 2);
}

__global__ void sha512_init_context_kernel(unsigned char *input, SHA512_CTX *ctx)
{

    SHA512_Init(ctx);
    SHA512_Update(ctx, input, 242);
}

long long timeInMilliseconds(void)
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

int main(int argc, char *argv[])
{
    //printf("%s %d\n", argv[1], argc);
    // Initialize context

    SHA512_CTX *sha_512_ctx = (SHA512_CTX *)malloc(sizeof(SHA512_CTX));
    SHA512_CTX *d_sha_512_ctx;
    hipMalloc(&d_sha_512_ctx, sizeof(SHA512_CTX));
    hipMemcpy(d_sha_512_ctx, sha_512_ctx, sizeof(SHA512_CTX), hipMemcpyHostToDevice);

    // Allocate memory on device for input and output
    unsigned char *d_input_p1;
    hipMalloc((void **)&d_input_p1, sizeof(char) * 243);
    hipMemcpy(d_input_p1, input_p1, sizeof(char) * 243, hipMemcpyHostToDevice);

    unsigned int *blockSolution = (unsigned int *)malloc(sizeof(unsigned int) * 2);
    unsigned int *d_solution;
    hipMalloc(&d_solution, sizeof(unsigned int) * 2);

    int *blockContainsSolution = (int *)malloc(sizeof(int));
    int *d_blockContainsSolution;
    hipMalloc(&d_blockContainsSolution, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(dev_K512), K512, sizeof(K512), 0, hipMemcpyHostToDevice);

    sha512_init_context_kernel<<<1, 1>>>(d_input_p1, d_sha_512_ctx);

    // hipDeviceSynchronize();

    unsigned char *d_output;
    hipMalloc((void **)&d_output, sizeof(char) * HASH_SIZE);

    // Copy input data from host to device

    unsigned char inputFromCmd[500];

    scanf("%500[^\n]", inputFromCmd);

    unsigned char hashedBytes[64];
    unsigned char completionTime[21];
    unsigned char serialNumber[8];

    int i;
    // Convert the hex-encoded SHA512 hashed string to a byte array
    for (i = 0; i < 64; i++)
    {
        char hex[3];
        hex[0] = inputFromCmd[2 * i];
        hex[1] = inputFromCmd[2 * i + 1];
        hex[2] = '\0';
        hashedBytes[i] = (uint8_t)strtol(hex, NULL, 16);
    }
    memcpy(completionTime, &inputFromCmd[129], sizeof(char) * 20);
    completionTime[20] = '\0';
    memcpy(serialNumber, &inputFromCmd[150], sizeof(char) * 7);
    serialNumber[7] = '\0';

    //printf("%s\n%s\n", completionTime, serialNumber);

    unsigned char *d_expectedHash;
    hipMalloc((void **)&d_expectedHash, sizeof(char) * 64);
    hipMemcpy(d_expectedHash, hashedBytes, sizeof(char) * 64, hipMemcpyHostToDevice);

    unsigned char *d_completionTime;
    hipMalloc((void **)&d_completionTime, sizeof(char) * 21);
    hipMemcpy(d_completionTime, completionTime, sizeof(char) * 21, hipMemcpyHostToDevice);

    unsigned char *d_serialNumber;
    hipMalloc((void **)&d_serialNumber, sizeof(char) * 8);
    hipMemcpy(d_serialNumber, serialNumber, sizeof(char) * 8, hipMemcpyHostToDevice);

    unsigned long hashCount = 0;
    long long start = timeInMilliseconds();

    unsigned int seed = 0;

    while (1)
    {
        hashCount += THREADS * BLOCKS;
        // Run SHA512 kernel
        sha512_kernel<<<THREADS, BLOCKS>>>(d_sha_512_ctx, seed, d_output, d_expectedHash, d_solution, d_blockContainsSolution, d_completionTime, d_serialNumber);

        hipDeviceSynchronize();

        hipMemcpy(blockContainsSolution, d_blockContainsSolution, sizeof(int), hipMemcpyDeviceToHost);
        if (*blockContainsSolution == 1)
        {
            hipMemcpy(blockSolution, d_solution, sizeof(unsigned int) * 2, hipMemcpyDeviceToHost);
            printf("%u,%u", blockSolution[0], blockSolution[1]);
            break;
        }

        //long long elapsed = timeInMilliseconds() - start;
        //printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\r", hashCount, ((float)elapsed) / 1000.0, (unsigned long)((double)hashCount / (double)elapsed) * 1000);
        seed++;
    }

    // Copy output data from device to host
    unsigned char output[HASH_SIZE];
    hipMemcpy(output, d_output, HASH_SIZE, hipMemcpyDeviceToHost);

    // Print output hash
    /* printf("Output hash: ");
    for (int i = 0; i < HASH_SIZE; i++)
    {
        printf("%02x", output[i]);
    }
    printf("\n"); */

    // Free memory on device
    hipFree(d_sha_512_ctx);
    hipFree(dev_K512);
    hipFree(d_expectedHash);
    hipFree(d_solution);
    hipFree(d_blockContainsSolution);
    hipFree(d_output);

    return 0;
}
